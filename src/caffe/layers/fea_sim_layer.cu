#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/fea_sim_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void FeaSimLayer<Dtype>::Forward_gpu(
  const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  int num = bottom[0]->shape(0);
  int spatial_dim = bottom[0]->count()/(bottom[0]->shape(0)*channels_);
  //image1 minus image2
  caffe_gpu_sub(
    count,
    bottom[0]->gpu_data(),
    bottom[1]->gpu_data(),
    diff_.mutable_gpu_data());
  //square the diff
  caffe_gpu_powx(
    count,
    diff_.gpu_data(),
    Dtype(2),
    pow_diff_.mutable_gpu_data());
  //diff divide by param -p^2
  caffe_gpu_powx(
    channels_,
    this->blobs_[0]->gpu_data(),
    Dtype(2),
    pow_p_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels_, 1, 1,
      batch_sum_multiplier_.gpu_data(), pow_p_.gpu_data(), 0.,
      num_by_chans_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_ * num,
      spatial_dim, 1, 1., num_by_chans_.gpu_data(),
      spatial_sum_multiplier_.gpu_data(), 0., temp_.mutable_gpu_data());
  caffe_gpu_div(
    count,
    pow_diff_.gpu_data(),
    temp_.gpu_data(),
    win_diff_.mutable_gpu_data());
  caffe_gpu_scale(
    count,
    Dtype(-1),
    win_diff_.gpu_data(),
    win_diff_.mutable_gpu_data());
  //make a exp
  caffe_gpu_exp(
    count,
    win_diff_.gpu_data(),
    top[0]->mutable_gpu_data());
  //LOG(INFO) << "top[0]->count: " << top[0]->count();
}

template <typename Dtype>
void FeaSimLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  int count = bottom[0]->count();
  int num = bottom[0]->shape(0);
  CHECK_GT(num, 0) << "height gets the error value";
  CHECK_GT(count, 0) << "count gets the error value";
  //LOG(INFO) << "the program had come here";
  int spatial_dim = bottom[0]->count()/(bottom[0]->shape(0)*channels_);
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels_, 1, 1,
      batch_sum_multiplier_.gpu_data(), this->blobs_[0]->gpu_data(), 0.,
      num_by_chans_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_ * num,
      spatial_dim, 1, 1., num_by_chans_.gpu_data(),
      spatial_sum_multiplier_.gpu_data(), 0., temp_p_.mutable_gpu_data());
  caffe_gpu_mul(count, top[0]->gpu_diff(), top[0]->gpu_data(), temp_.mutable_gpu_data());
  caffe_gpu_div(count, temp_.gpu_data(), temp_p_.gpu_data(), temp_.mutable_gpu_data());
  caffe_gpu_scale(count, Dtype(2), temp_.gpu_data(), temp_.mutable_gpu_data());
  //bottom[0]_diff
  caffe_gpu_sub(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(), temp_sub_.mutable_gpu_data());
  caffe_gpu_mul(count, temp_.gpu_data(), temp_sub_.gpu_data(), bottom[0]->mutable_gpu_diff());
  //bottom[1]_diff
  caffe_gpu_sub(count, bottom[1]->gpu_data(), bottom[0]->gpu_data(), temp_sub_.mutable_gpu_data());
  caffe_gpu_mul(count, temp_.gpu_data(), temp_sub_.gpu_data(), bottom[1]->mutable_gpu_diff());
  //blob_diff
    //bottom[0]-bottom[1] pow2
  caffe_gpu_sub(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(), temp_.mutable_gpu_data());
  caffe_gpu_powx(count, temp_.gpu_data(), Dtype(2), temp_.mutable_gpu_data());
    //blob[0]pow2
  caffe_gpu_powx(count, temp_p_.gpu_data(), Dtype(3), temp_p_.mutable_gpu_data());
    //blob[0]diff
  caffe_gpu_mul(count, top[0]->gpu_diff(), top[0]->gpu_data(), temp_blob_.mutable_gpu_data());
  caffe_gpu_mul(count, temp_blob_.gpu_data(), temp_.gpu_data(), temp_blob_.mutable_gpu_data());
  caffe_gpu_div(count, temp_blob_.gpu_data(), temp_p_.gpu_data(), temp_blob_.mutable_gpu_data());
  caffe_gpu_scale(count, Dtype(2), temp_blob_.gpu_data(), temp_blob_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num * channels_, 1, spatial_dim ,1.,
    temp_blob_.gpu_data(), spatial_sum_multiplier_.gpu_data(), 0., num_by_chans_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, 1, channels_, num, 1.,
    batch_sum_multiplier_.gpu_data(), num_by_chans_.gpu_data(), 0., this->blobs_[0]->mutable_gpu_diff());
  //LOG(INFO) << "the program had come here";
}
INSTANTIATE_LAYER_GPU_FUNCS(FeaSimLayer);

}  // namespace caffe